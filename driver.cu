#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define MIN_EPSILON_ERROR 5e-3f

////////////////////////////////////////////////////////////////////////////////
// Define the files that are to be save and the reference images for validation
const char *imageFilename = "lena.ppm";
//const char *refFilename   = "ref_rotated.pgm";

// Auto-Verification Code
bool testResult = true;
